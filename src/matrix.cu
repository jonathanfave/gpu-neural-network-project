#include "hip/hip_runtime.h"
#include "matrix.h"
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

matrix_t * alloc_matrix(unsigned rows, unsigned columns)
{
    matrix_t * res = (matrix_t*) malloc( sizeof(matrix_t) );
    res->m = (double *) calloc(columns * rows, sizeof(double));
    res->columns = columns;
    res->rows = rows;
    return res;
}

void destroy_matrix(matrix_t *m)
{
    //printf("free %p %p\n", m, m->m);
    free(m->m);
    free(m);
}

void print_matrix(matrix_t *m, bool is_short){
    unsigned lim_rows = 0;
    unsigned lim_col = 0;

    if (is_short)
    {
        lim_rows = MIN(m->rows, 4);
        lim_col = MIN(m->columns, 10);
    }
    else
    {
        lim_rows = m->rows;
        lim_col = m->columns;
    }

    for (int row = 0; row < lim_rows; row ++)
    {
        for (int col = 0; col < lim_col; col ++)
        {
            printf("%.2lf ", m->m[col + row * m->columns]);
        }
        if (is_short && lim_col != m->columns) printf("...");
        printf("\n");
    }
    if (is_short && lim_rows != m->rows) printf("...\n");
}

void hadamard_product(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)   &&
             (m1->columns == res->columns)  &&
             (m1->rows == m2->rows)         &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
            res->m[idx] = m1->m[idx] * m2->m[idx];
    }
}

void matrix_sum(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    { 
        res->m[idx] = m1->m[idx] + m2->m[idx];
    }
}

void matrix_minus(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));
             
    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = m1->m[idx] - m2->m[idx];
    }
}

// Kernel for matrix multiplication
__global__ 
void matrixMultiplyKernel(double *A, double *B, double *res, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * k + col];
        }
        res[row * k + col] = sum;
    }
}

// Function to perform matrix multiplication using CUDA with unified memory
void matrixDotCUDAManaged(double *h_A, double *h_B, double *h_res, int m, int n, int k) {
    // Allocate unified memory
    double *d_A, *d_B, *d_res;
    hipMallocManaged(&d_A, m * n * sizeof(double));
    hipMallocManaged(&d_B, n * k * sizeof(double));
    hipMallocManaged(&d_res, m * k * sizeof(double));

    // Transfer data to unified memory
    hipMemcpy(d_A, h_A, m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * k * sizeof(double), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 dimBlock(16, 16);
    dim3 dimGrid((k + dimBlock.x - 1) / dimBlock.x, (m + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_res, m, n, k);
    hipDeviceSynchronize(); // Wait for kernel to finish execution

    // Transfer result back to host
    hipMemcpy(h_res, d_res, m * k * sizeof(double), hipMemcpyDeviceToHost);

    // Free unified memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_res);
}


// Function to perform matrix multiplication using CUDA
void matrixDotCUDA(double *h_A, double *h_B, double *h_res, int m, int n, int k) {
    int size_A = m * n * sizeof(float);
    int size_B = n * k * sizeof(float);
    int size_res = m * k * sizeof(float);

    // Allocate memory on the GPU
    double *d_A, *d_B, *d_res;
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_res, size_res);

    // Transfer data from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 dimBlock(16, 16);
    dim3 dimGrid((k + dimBlock.x - 1) / dimBlock.x, (m + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_res, m, n, k);

    // Transfer result from device to host
    hipMemcpy(h_res, d_res, size_res, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_res);
}

// Existing matrix_dot function
void matrix_dot(double *A, double *B, double *res, int m, int n, int k) {
    // Call the optimized CUDA function
    matrixDotCUDA(A, B, res, n, m, k);
}

void matrix_dot(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->rows)  &&
             (m1->rows == res->rows)    &&
             (m2->columns == res->columns));

    for (int row = 0; row < m1->rows; row ++)
    {   
        for (int col = 0; col < m2->columns; col ++)
        {
            int idx = col + row * m2->columns;
            double var = 0.0;

            for (int ii = 0; ii < m1->columns; ii++)
            {
                var += m1->m[ii + row * m1->columns] * m2->m[col + ii * m2->columns];
            }

            res->m[idx] = var;
        }
    }
}

void matrix_function(matrix_t *m1, double (*f)(double), matrix_t *res)
{
    assert ( (m1->columns == res->columns) &&             
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = f(m1->m[idx]);
    }
}

void matrix_transpose(matrix_t *m1, matrix_t *res)
{
    assert ( (m1->columns == res->rows) &&             
             (m1->rows == res->columns));
    
    for (int row = 0; row < m1->rows; row++)
    {
        for (int col = 0; col < m1->columns; col ++)
        {
            res->m[row + col * m1->rows] = m1->m[col + row * m1->columns];
        }
    }
}

void matrix_scalar(matrix_t *m1, double s, matrix_t *res)
{
    assert ( (m1->rows == res->rows) &&             
             (m1->columns == res->columns));

    for (int idx = 0; idx < m1->columns*m1->rows; idx ++)
    {
        res->m[idx] = m1->m[idx] * s;
    }
}

void matrix_memcpy(matrix_t *dest, const matrix_t *src)
{
    assert ( (dest->rows == src->rows)      &&             
             (dest->columns == src->columns));

    memcpy(dest->m, src->m, src->columns * src->rows * sizeof(double));     
}